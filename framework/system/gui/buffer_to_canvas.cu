#include "hip/hip_runtime.h"
#include "buffer_to_canvas.cuh"
#include "cuda/kernel.h"

namespace Pupil {
    void CopyFromFloat1(hipDeviceptr_t dst, hipDeviceptr_t src, unsigned int size, cuda::Stream* stream) noexcept {
        Pupil::cuda::LaunchKernel1D(
            size, [dst, src] __device__(unsigned int index, unsigned int size) {
                auto output   = reinterpret_cast<float4*>(dst);
                auto input    = reinterpret_cast<float*>(src);
                output[index] = make_float4(input[index], input[index], input[index], 1.f);
            },
            stream);
    }

    void CopyFromFloat2(hipDeviceptr_t dst, hipDeviceptr_t src, unsigned int size, cuda::Stream* stream) noexcept {
        Pupil::cuda::LaunchKernel1D(
            size, [dst, src] __device__(unsigned int index, unsigned int size) {
                auto output   = reinterpret_cast<float4*>(dst);
                auto input    = reinterpret_cast<float2*>(src);
                output[index] = make_float4(input[index].x, input[index].y, 0.f, 1.f);
            },
            stream);
    }

    void CopyFromFloat3(hipDeviceptr_t dst, hipDeviceptr_t src, unsigned int size, cuda::Stream* stream) noexcept {
        Pupil::cuda::LaunchKernel1D(
            size, [dst, src] __device__(unsigned int index, unsigned int size) {
                auto output   = reinterpret_cast<float4*>(dst);
                auto input    = reinterpret_cast<float3*>(src);
                output[index] = make_float4(input[index].x, input[index].y, input[index].z, 1.f);
            },
            stream);
    }
}// namespace Pupil