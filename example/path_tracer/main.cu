#include "hip/hip_runtime.h"
#include <optix.h>
#include "type.h"

#include "optix/util.h"
#include "render/geometry.h"
#include "render/material/bsdf/bsdf.h"

#include "cuda/random.h"

using namespace Pupil;

extern "C" {
__constant__ pt::OptixLaunchParams optix_launch_params;
}

struct HitInfo {
    optix::LocalGeometry geo;
    optix::material::Material::LocalBsdf bsdf;
    int emitter_index;
};

struct PathPayloadRecord {
    float3 radiance;
    float3 env_radiance;
    float env_pdf;
    cuda::Random random;

    float3 throughput;

    HitInfo hit;

    unsigned int depth;
    bool done;

    float test;
};

extern "C" __global__ void __raygen__main() {
    const uint3 index = optixGetLaunchIndex();
    const unsigned int w = optix_launch_params.config.frame.width;
    const unsigned int h = optix_launch_params.config.frame.height;
    const unsigned int pixel_index = index.y * w + index.x;

    auto &camera = *optix_launch_params.camera.GetDataPtr();

    PathPayloadRecord record{};
    uint32_t u0, u1;
    optix::PackPointer(&record, u0, u1);

    record.done = false;
    record.depth = 0u;
    record.throughput = make_float3(1.f);
    record.radiance = make_float3(0.f);
    record.env_radiance = make_float3(0.f);
    record.random.Init(4, pixel_index, optix_launch_params.random_seed);
    record.test = 0.f;

    const float2 subpixel_jitter = make_float2(record.random.Next(), record.random.Next());

    const float2 subpixel =
        make_float2(
            (static_cast<float>(index.x) + subpixel_jitter.x) / static_cast<float>(w),
            (static_cast<float>(index.y) + subpixel_jitter.y) / static_cast<float>(h));
    // const float2 subpixel = make_float2((static_cast<float>(index.x)) / w, (static_cast<float>(index.y)) / h);
    const float4 point_on_film = make_float4(subpixel, 0.f, 1.f);

    float4 d = camera.sample_to_camera * point_on_film;

    d /= d.w;
    d.w = 0.f;
    d = normalize(d);

    float3 ray_direction = normalize(make_float3(camera.camera_to_world * d));

    float3 ray_origin = make_float3(
        camera.camera_to_world.r0.w,
        camera.camera_to_world.r1.w,
        camera.camera_to_world.r2.w);

    optixTrace(optix_launch_params.handle,
               ray_origin, ray_direction,
               0.001f, 1e16f, 0.f,
               255, OPTIX_RAY_FLAG_NONE,
               0, 2, 0,
               u0, u1);

    int depth = 0;
    auto local_hit = record.hit;

    if (!record.done) {
        if (record.hit.emitter_index >= 0) {
            auto &emitter = optix_launch_params.emitters.areas[local_hit.emitter_index];
            auto emission = emitter.GetRadiance(local_hit.geo.texcoord);
            record.radiance += emission;
        }

        optix_launch_params.albedo_buffer[pixel_index] = local_hit.bsdf.GetAlbedo();
        optix_launch_params.normal_buffer[pixel_index] = local_hit.geo.normal;
    } else {
        optix_launch_params.albedo_buffer[pixel_index] = make_float3(0.f);
        optix_launch_params.normal_buffer[pixel_index] = make_float3(0.f);
    }

    optix_launch_params.test[pixel_index] = record.random.Next();

    while (!record.done) {
        ++depth;
        if (depth >= optix_launch_params.config.max_depth)
            break;

        float rr = depth > 2 ? 0.95 : 1.0;
        if (record.random.Next() > rr)
            break;
        record.throughput /= rr;

        // direct light sampling
        {
            auto &emitter = optix_launch_params.emitters.SelectOneEmiiter(record.random.Next());
            optix::EmitterSampleRecord emitter_sample_record;
            emitter.SampleDirect(emitter_sample_record, local_hit.geo, record.random.Next2());

            bool occluded =
                optix::Emitter::TraceShadowRay(
                    optix_launch_params.handle,
                    local_hit.geo.position, emitter_sample_record.wi,
                    0.0001f, emitter_sample_record.distance - 0.0001f);
            if (!occluded) {
                optix::BsdfSamplingRecord eval_record;
                eval_record.wi = optix::ToLocal(emitter_sample_record.wi, local_hit.geo.normal);
                eval_record.wo = optix::ToLocal(-ray_direction, local_hit.geo.normal);
                eval_record.sampler = &record.random;
                record.hit.bsdf.Eval(eval_record);
                float3 f = eval_record.f;
                float pdf = eval_record.pdf;
                if (!optix::IsZero(f * emitter_sample_record.pdf)) {
                    float NoL = dot(local_hit.geo.normal, emitter_sample_record.wi);
                    if (NoL > 0.f) {
                        float mis = emitter_sample_record.is_delta ? 1.f : optix::MISWeight(emitter_sample_record.pdf, pdf);
                        emitter_sample_record.pdf *= emitter.select_probability;
                        record.radiance += record.throughput * emitter_sample_record.radiance * f * NoL * mis / emitter_sample_record.pdf;
                    }
                }
            }
        }
        // bsdf sampling
        {
            float3 wo = optix::ToLocal(-ray_direction, local_hit.geo.normal);
            optix::BsdfSamplingRecord bsdf_sample_record;
            bsdf_sample_record.wo = optix::ToLocal(-ray_direction, local_hit.geo.normal);
            bsdf_sample_record.sampler = &record.random;
            record.hit.bsdf.Sample(bsdf_sample_record);

            if (optix::IsZero(bsdf_sample_record.f * abs(bsdf_sample_record.wi.z)) || optix::IsZero(bsdf_sample_record.pdf))
                break;

            record.throughput *= bsdf_sample_record.f * abs(bsdf_sample_record.wi.z) / bsdf_sample_record.pdf;

            ray_origin = record.hit.geo.position;
            ray_direction = optix::ToWorld(bsdf_sample_record.wi, local_hit.geo.normal);

            optixTrace(optix_launch_params.handle,
                       ray_origin, ray_direction,
                       0.001f, 1e16f, 0.f,
                       255, OPTIX_RAY_FLAG_NONE,
                       0, 2, 0,
                       u0, u1);

            if (record.done) {
                float mis = optix::MISWeight(bsdf_sample_record.pdf, record.env_pdf);
                record.env_radiance *= record.throughput * mis;
                break;
            }

            local_hit = record.hit;
            if (record.hit.emitter_index >= 0) {
                auto &emitter = optix_launch_params.emitters.areas[record.hit.emitter_index];
                optix::EmitEvalRecord emit_record;
                emitter.Eval(emit_record, record.hit.geo, ray_origin);
                if (!optix::IsZero(emit_record.pdf)) {
                    float mis = bsdf_sample_record.sampled_type & optix::EBsdfLobeType::Delta ?
                                    1.f :
                                    optix::MISWeight(bsdf_sample_record.pdf, emit_record.pdf * emitter.select_probability);
                    record.radiance += record.throughput * emit_record.radiance * mis;
                }
            }
        }
    }
    record.radiance += record.env_radiance;

    if (optix_launch_params.config.accumulated_flag && optix_launch_params.sample_cnt > 0) {
        const float t = 1.f / (optix_launch_params.sample_cnt + 1.f);
        const float3 pre = make_float3(optix_launch_params.accum_buffer[pixel_index]);
        record.radiance = lerp(pre, record.radiance, t);
    }
    optix_launch_params.accum_buffer[pixel_index] = make_float4(record.radiance, 1.f);
    optix_launch_params.frame_buffer[pixel_index] = make_float4(record.radiance, 1.f);
}

extern "C" __global__ void __miss__default() {
    auto record = optix::GetPRD<PathPayloadRecord>();
    if (optix_launch_params.emitters.env) {
        auto &env = *optix_launch_params.emitters.env.GetDataPtr();

        const auto ray_dir = normalize(optixGetWorldRayDirection());
        const auto ray_o = optixGetWorldRayOrigin();

        optix::LocalGeometry env_local;
        env_local.position = ray_o + ray_dir;
        optix::EmitEvalRecord emit_record;
        env.Eval(emit_record, env_local, ray_o);
        record->env_radiance = emit_record.radiance;
        record->env_pdf = emit_record.pdf;
    }
    record->done = true;
}
extern "C" __global__ void __miss__shadow() {
    // optixSetPayload_0(0u);
}

__device__ __forceinline__ void ClosestHit() {
    const pt::HitGroupData *sbt_data = (pt::HitGroupData *)optixGetSbtDataPointer();
    auto record = optix::GetPRD<PathPayloadRecord>();

    const auto ray_dir = optixGetWorldRayDirection();
    const auto ray_o = optixGetWorldRayOrigin();

    sbt_data->geo.GetHitLocalGeometry(record->hit.geo, ray_dir, sbt_data->mat.twosided);
    if (sbt_data->emitter_index_offset >= 0) {
        record->hit.emitter_index = sbt_data->emitter_index_offset + optixGetPrimitiveIndex();
    } else {
        record->hit.emitter_index = -1;
    }
    record->hit.bsdf = sbt_data->mat.GetLocalBsdf(record->hit.geo.texcoord);
}

__device__ __forceinline__ void ClosestHitShadow() {
    optixSetPayload_0(1u);
}

extern "C" __global__ void __closesthit__default() { ClosestHit(); }
extern "C" __global__ void __closesthit__default_sphere() { ClosestHit(); }
extern "C" __global__ void __closesthit__default_curve() { ClosestHit(); }

extern "C" __global__ void __closesthit__shadow() { ClosestHitShadow(); }
extern "C" __global__ void __closesthit__shadow_sphere() { ClosestHitShadow(); }
extern "C" __global__ void __closesthit__shadow_curve() { ClosestHitShadow(); }